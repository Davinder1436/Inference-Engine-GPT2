#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "common.h"

// Ultra-simple test to isolate the attention memory issue
__global__ void test_simple_copy(float* out, float* inp, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = inp[idx];
    }
}

int main() {
    const int B = 1, NH = 12, T = 1;  // Start with the simplest case
    const int size = B * NH * T * T;
    
    float *d_in, *d_out;
    hipMalloc(&d_in, size * sizeof(float));
    hipMalloc(&d_out, size * sizeof(float));
    
    // Initialize with simple values
    float init_val = 1.0f;
    hipMemset(d_in, 0, size * sizeof(float));
    hipMemcpy(d_in, &init_val, sizeof(float), hipMemcpyHostToDevice);
    
    // Test simple copy
    int block_size = 256;
    int grid_size = (size + block_size - 1) / block_size;
    test_simple_copy<<<grid_size, block_size>>>(d_out, d_in, size);
    
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    
    std::cout << "Simple copy test passed!" << std::endl;
    
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}
