#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define checkCuda(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA Error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
} while(0)

// Debug function to test each step of attention computation
void debug_attention_step_by_step() {
    std::cout << "=== ATTENTION DEBUG MODE ===" << std::endl;
    
    // Test parameters (same as in main)
    int B = 1, T = 1, C = 768, NH = 12;
    int HS = C / NH;  // 64
    
    std::cout << "Parameters: B=" << B << ", T=" << T << ", C=" << C << ", NH=" << NH << ", HS=" << HS << std::endl;
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    // === STEP 1: Test memory allocations ===
    std::cout << "\n1. Testing memory allocations..." << std::endl;
    float *d_qkv, *d_q, *d_k, *d_v, *d_att, *d_att_softmax, *d_y;
    float *d_input;
    
    try {
        checkCuda(hipMalloc(&d_input, B * T * C * sizeof(float)));
        checkCuda(hipMalloc(&d_qkv, B * T * 3 * C * sizeof(float)));
        checkCuda(hipMalloc(&d_q, B * NH * T * HS * sizeof(float)));
        checkCuda(hipMalloc(&d_k, B * NH * T * HS * sizeof(float)));
        checkCuda(hipMalloc(&d_v, B * NH * T * HS * sizeof(float)));
        checkCuda(hipMalloc(&d_att, B * NH * T * T * sizeof(float)));
        checkCuda(hipMalloc(&d_att_softmax, B * NH * T * T * sizeof(float)));
        checkCuda(hipMalloc(&d_y, B * T * C * sizeof(float)));
        std::cout << "   ✓ All memory allocations successful" << std::endl;
    } catch (...) {
        std::cout << "   ✗ Memory allocation failed!" << std::endl;
        return;
    }
    
    // === STEP 2: Initialize with safe values ===
    std::cout << "\n2. Initializing with test data..." << std::endl;
    checkCuda(hipMemset(d_input, 0, B * T * C * sizeof(float)));
    checkCuda(hipMemset(d_qkv, 0, B * T * 3 * C * sizeof(float)));
    checkCuda(hipMemset(d_att, 0, B * NH * T * T * sizeof(float)));
    
    // Set some test values
    float test_val = 1.0f;
    checkCuda(hipMemcpy(d_input, &test_val, sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_att, &test_val, sizeof(float), hipMemcpyHostToDevice));
    std::cout << "   ✓ Test data initialized" << std::endl;
    
    // === STEP 3: Test simple memory copy ===
    std::cout << "\n3. Testing simple memory operations..." << std::endl;
    checkCuda(hipMemcpy(d_att_softmax, d_att, B * NH * T * T * sizeof(float), hipMemcpyDeviceToDevice));
    std::cout << "   ✓ Device-to-device copy successful" << std::endl;
    
    // Test CPU-GPU copy
    float* h_test = new float[B * NH * T * T];
    checkCuda(hipMemcpy(h_test, d_att, B * NH * T * T * sizeof(float), hipMemcpyDeviceToHost));
    std::cout << "   ✓ GPU-to-CPU copy successful, first value: " << h_test[0] << std::endl;
    delete[] h_test;
    
    // === STEP 4: Test each attention component individually ===
    std::cout << "\n4. Testing attention components..." << std::endl;
    
    // Test QKV computation (this might be where cuBLAS fails)
    std::cout << "   4a. Testing QKV projection..." << std::endl;
    // We'll skip this since we don't have weights loaded
    
    // Test attention matrix computation
    std::cout << "   4b. Testing attention matrix..." << std::endl;
    // Initialize Q, K with simple values
    checkCuda(hipMemset(d_q, 0, B * NH * T * HS * sizeof(float)));
    checkCuda(hipMemset(d_k, 0, B * NH * T * HS * sizeof(float)));
    
    test_val = 0.1f;
    checkCuda(hipMemcpy(d_q, &test_val, sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_k, &test_val, sizeof(float), hipMemcpyHostToDevice));
    
    // Test cuBLAS batched matrix multiplication
    const float** q_ptr_array = new const float*[B * NH];
    const float** k_ptr_array = new const float*[B * NH];
    float** att_ptr_array = new float*[B * NH];
    
    for(int i = 0; i < B * NH; ++i) {
        q_ptr_array[i] = d_q + i * T * HS;
        k_ptr_array[i] = d_k + i * T * HS;
        att_ptr_array[i] = d_att + i * T * T;
    }
    
    float alpha = 1.0f, beta = 0.0f;
    hipblasStatus_t cublas_status = hipblasSgemmBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, T, T, HS, 
                                                      &alpha, k_ptr_array, HS, q_ptr_array, HS, 
                                                      &beta, att_ptr_array, T, B * NH);
    
    if (cublas_status == HIPBLAS_STATUS_SUCCESS) {
        std::cout << "   ✓ cuBLAS batched operation successful" << std::endl;
    } else {
        std::cout << "   ✗ cuBLAS batched operation failed with status: " << cublas_status << std::endl;
    }
    
    delete[] q_ptr_array;
    delete[] k_ptr_array;
    delete[] att_ptr_array;
    
    // === STEP 5: Test the problematic softmax step ===
    std::cout << "\n5. Testing softmax step (the failing part)..." << std::endl;
    
    // First, verify the attention matrix is valid
    float* h_att_check = new float[B * NH * T * T];
    hipError_t copy_err = hipMemcpy(h_att_check, d_att, B * NH * T * T * sizeof(float), hipMemcpyDeviceToHost);
    
    if (copy_err == hipSuccess) {
        std::cout << "   ✓ Attention matrix is readable from GPU" << std::endl;
        std::cout << "   First few values: ";
        for (int i = 0; i < std::min(5, B * NH * T * T); i++) {
            std::cout << h_att_check[i] << " ";
        }
        std::cout << std::endl;
        
        // Now test our attention_softmax function with valid data
        std::cout << "   Testing attention_softmax with valid data..." << std::endl;
        // This is where we'd call attention_softmax_forward, but we know it fails
        
    } else {
        std::cout << "   ✗ Cannot read attention matrix from GPU: " << hipGetErrorString(copy_err) << std::endl;
    }
    
    delete[] h_att_check;
    
    // === Cleanup ===
    hipFree(d_input);
    hipFree(d_qkv);
    hipFree(d_q);
    hipFree(d_k);
    hipFree(d_v);
    hipFree(d_att);
    hipFree(d_att_softmax);
    hipFree(d_y);
    hipblasDestroy(handle);
    
    std::cout << "\n=== DEBUG COMPLETE ===" << std::endl;
}

int main() {
    debug_attention_step_by_step();
    return 0;
}
