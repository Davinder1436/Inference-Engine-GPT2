#include "linear.h"

void linear_forward(float* out, float* inp, float* weight, hipblasHandle_t handle, int B, int T, int C, int OC) {
    float alpha = 1.0f, beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, OC, B * T, C, &alpha, weight, C, inp, C, &beta, out, OC);
}
