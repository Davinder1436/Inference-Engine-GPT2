#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define checkCuda(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA Error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
} while(0)

int main() {
    std::cout << "=== ATTENTION STEP-BY-STEP DEBUG ===" << std::endl;
    
    // Test parameters (same as in main)
    int B = 1, T = 1, C = 768, NH = 12;
    int HS = C / NH;
    
    std::cout << "Parameters: B=" << B << ", T=" << T << ", C=" << C << ", NH=" << NH << ", HS=" << HS << std::endl;
    
    // Test 1: Basic memory allocation
    std::cout << "\n1. Testing basic memory allocation..." << std::endl;
    float *d_att, *d_att_softmax;
    int att_size = B * NH * T * T;
    
    checkCuda(hipMalloc(&d_att, att_size * sizeof(float)));
    checkCuda(hipMalloc(&d_att_softmax, att_size * sizeof(float)));
    std::cout << "   ✓ Memory allocated successfully (" << att_size << " elements)" << std::endl;
    
    // Test 2: Memory initialization
    std::cout << "\n2. Testing memory initialization..." << std::endl;
    checkCuda(hipMemset(d_att, 0, att_size * sizeof(float)));
    
    float test_val = 1.0f;
    checkCuda(hipMemcpy(d_att, &test_val, sizeof(float), hipMemcpyHostToDevice));
    std::cout << "   ✓ Memory initialized successfully" << std::endl;
    
    // Test 3: Memory read back
    std::cout << "\n3. Testing memory read back..." << std::endl;
    float* h_test = new float[att_size];
    checkCuda(hipMemcpy(h_test, d_att, att_size * sizeof(float), hipMemcpyDeviceToHost));
    std::cout << "   ✓ Memory read back successful, first value: " << h_test[0] << std::endl;
    
    // Test 4: cuBLAS setup (this might be the culprit)
    std::cout << "\n4. Testing cuBLAS operations..." << std::endl;
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status == HIPBLAS_STATUS_SUCCESS) {
        std::cout << "   ✓ cuBLAS handle created successfully" << std::endl;
        
        // Test a simple cuBLAS operation
        float *d_a, *d_b, *d_c;
        int n = 64;  // HS size
        checkCuda(hipMalloc(&d_a, n * n * sizeof(float)));
        checkCuda(hipMalloc(&d_b, n * n * sizeof(float)));
        checkCuda(hipMalloc(&d_c, n * n * sizeof(float)));
        
        // Initialize with small values
        checkCuda(hipMemset(d_a, 0, n * n * sizeof(float)));
        checkCuda(hipMemset(d_b, 0, n * n * sizeof(float)));
        test_val = 0.1f;
        checkCuda(hipMemcpy(d_a, &test_val, sizeof(float), hipMemcpyHostToDevice));
        checkCuda(hipMemcpy(d_b, &test_val, sizeof(float), hipMemcpyHostToDevice));
        
        // Simple matrix multiplication
        float alpha = 1.0f, beta = 0.0f;
        status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n);
        
        if (status == HIPBLAS_STATUS_SUCCESS) {
            std::cout << "   ✓ Basic cuBLAS operation successful" << std::endl;
        } else {
            std::cout << "   ✗ Basic cuBLAS operation failed: " << status << std::endl;
        }
        
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        hipblasDestroy(handle);
    } else {
        std::cout << "   ✗ cuBLAS handle creation failed: " << status << std::endl;
    }
    
    delete[] h_test;
    hipFree(d_att);
    hipFree(d_att_softmax);
    
    std::cout << "\n=== All basic tests passed! The issue is likely in the attention computation logic ===" << std::endl;
    return 0;
}
